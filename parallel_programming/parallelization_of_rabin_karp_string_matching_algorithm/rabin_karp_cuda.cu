/**
 * @file rabin_karp_cuda.cu
 * @brief 
 * @version 0.1
 * @date 2022-08-03
 * 
 * @copyright Copyright (c) 2022
 * 
 */

#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include "math.h"
#include "time.h"
#include "float.h"
#include "cstdio"

#define N 1048576
#define N_BLOCKS 1024
#define BLOCK_SIZE 1024

__device__ float counts[N_BLOCKS * BLOCK_SIZE];

/**
 * @brief 
 * 
 */
__global__ void rabin_karp(char *text, char *pattern, int text_len, int pat_len, int chunk_size)
{
    int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    int i;
    int j;

    int base = 10;
    int divisor = 17;

    int ph = 0; //pattern hash
    int th = 0; // text hash
    int bh = 1; // base hash
    int start = threadID * chunk_size;

    counts[threadID] = 0;

    __syncthreads();

    if(start + chunk_size + pat_len > text_len) {
        return;
    }
    
    // Get hash value of bases of any (m-1)-length-text. Used in rolling hash calculation
    for (i = 0; i < pat_len - 1; i++)
        bh = (bh * base) % divisor;

    // Get hash value for pattern and m-length-text at the beginning of the text
    for (i = 0; i < pat_len; i++) {
        ph = (base * ph + pattern[i]) % divisor;
        th = (base * th + text[i + start]) % divisor;
    }

    // Find the match
    for (i = start; i < start + chunk_size; i++) {//loop1 START
        if (ph == th) {
            for (j = 0; j < pat_len; j++) {
                if (text[i + j] != pattern[j])
                break;
            }

            if (j == pat_len){//j will be equal to pat_len if all characters match
                // printf("Match at position:  %d \n", i + 1);
                counts[threadID] = counts[threadID] + 1;
            }
        }

        // Rolling hash calculation
        if (i < text_len - pat_len) {
            th = (base * (th - text[i] * bh) + text[i + pat_len]) % divisor;
            if (th < 0)
                th = (th + divisor);
        }
    }//loop1 END

    return;
}

__global__ void init_kernel(int *count)
{
    for(int i = 0; i < N_BLOCKS * BLOCK_SIZE; i++) {
        counts[i] = 0;
    }
    *count = 0;
    return;
}

// First step in binary reduction:
__global__ void reduction1 (int *summed_counts)
{
    __shared__ int local_counts[BLOCK_SIZE];

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    local_counts[threadIdx.x] = counts[i];

    // To make sure all threads in a block have the count[] value:
    __syncthreads();

    int nTotalThreads = blockDim.x;  // Total number of active threads;
    // only the first half of the threads will be active.

    while(nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads >> 1);     // divide by two

        if (threadIdx.x < halfPoint)
        {
            int thread2 = threadIdx.x + halfPoint;
            local_counts[threadIdx.x] = local_counts[threadIdx.x] + local_counts[thread2];
        }   
        __syncthreads();
        nTotalThreads = halfPoint;  // Reducing the binary tree size by two
    }

    if (threadIdx.x == 0)
    {
        summed_counts[blockIdx.x] = local_counts[0];
    }

    return;
}



// Second step in binary reduction (one block):
__global__ void reduction2 (int *summed_counts, int *count)
{
    __shared__ int local_counts[N_BLOCKS];

    // Copying from global to shared memory:
    local_counts[threadIdx.x] = summed_counts[threadIdx.x];

    // To make sure all threads in a block have the min[] value:
    __syncthreads();

    int nTotalThreads = blockDim.x;  // Total number of active threads;
    // only the first half of the threads will be active.

    while(nTotalThreads > 1)
    {
        int halfPoint = (nTotalThreads >> 1);     // divide by two

        if (threadIdx.x < halfPoint)
            {
            int thread2 = threadIdx.x + halfPoint;
            local_counts[threadIdx.x] = local_counts[threadIdx.x] + local_counts[thread2];
        }   
        __syncthreads();
        nTotalThreads = halfPoint;  // Reducing the binary tree size by two
    }

    if (threadIdx.x == 0)
    {
        *count = local_counts[0];
    }

    return;
}

/**
 * @brief 
 * 
 * @param argc 
 * @param argv 
 * @return int 
 */
int main(int argc, char *argv[])
{
    int total_count = 0;
    int *host_count;
    char *text_dev;    /* array for computation on device */
    char *pattern_dev;
    //char *text = "The Wilfrid Laurier University or the University of Wilfrid Laurier";
    char *pattern;
    int *summed_counts;
    int *count;
    char *text;
    int num_chunks = N_BLOCKS * BLOCK_SIZE;
    long total_chars_processed = 0;
    
    int chunksize;
    size_t memsize;
    size_t patsize;

    text = (char *)malloc(N * sizeof(char));
    pattern = (char *)malloc(4 * sizeof(char));

    FILE *fp;

    fp = fopen("data/windows_mb512.log", "rb");
    fseek(fp, 0L, SEEK_END);
    size_t file_size = ftell(fp);
    printf("file size: %d\n", file_size);
    rewind(fp);

    // seed random number generator
    //srand(time(NULL));

    //printf("generating random characters\n");

    /*
    for(int i = 0; i < N; i++) {
        text[i] = (char) ('A' + (rand() % 4));
    }

    printf("finished generating characters\n");
    
    printf("text: ");
    printf(text);
    printf("\n");
    printf("pattern: ");
    printf(pattern);
    printf("\n");
    */

    pattern[0] = 'F';
    pattern[1] = 'a';
    pattern[2] = 'i';
    pattern[3] = 'l';
    pattern[4] = 'e';
    pattern[5] = 'd';

    // assuming the length of the string is evenly divided by the block size and
    // number

    host_count = (int *)malloc(sizeof(int));

    /* allocate arrays on device */

    hipMalloc((void **) &text_dev, memsize);
    hipMalloc((void **) &pattern_dev, patsize);
    hipMalloc((void **) &summed_counts, N_BLOCKS * sizeof(int));
    hipMalloc((void **) &count, sizeof(int));
     
    clock_t start, end;
    double cpu_time_used;

    start = clock();

    for( int i = 0; i < file_size; i += N) {

        fread(text, N, sizeof(char), fp);

        chunksize = strlen(text) / num_chunks;
        memsize = strlen(text) * sizeof(char);
        patsize = strlen(pattern) * sizeof(char);

        printf("text size: %d \n", memsize);
        printf("pattern size: %d \n", patsize);
        printf("chunk size: %d \n", chunksize);

        printf("Copying text to device\n");
        /* copy arrays to device memory (synchronous) */
        hipMemcpy(text_dev, text, memsize, hipMemcpyHostToDevice);
        hipMemcpy(pattern_dev, pattern, patsize, hipMemcpyHostToDevice);

        hipDeviceSynchronize();

        printf("Initializing Kernel\n");
        init_kernel<<<1, 1>>> (count);

        printf("Executing Pattern Matching\n");
        rabin_karp <<< N_BLOCKS, BLOCK_SIZE >>> (text_dev, pattern_dev, memsize, patsize, chunksize);

        hipDeviceSynchronize();

        printf("Executing first level of reduction\n");
        // First level binary reduction:
        reduction1 <<< N_BLOCKS, BLOCK_SIZE >>> (summed_counts);

        hipDeviceSynchronize();

        printf("Executing second level of reduction\n");
        // Second level binary reduction (only one block):
        reduction2 <<< 1, N_BLOCKS >>> (summed_counts, count);

        hipDeviceSynchronize();

        hipMemcpy(host_count, count, sizeof(int), hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        total_chars_processed += N;
        total_count += *host_count;
        // print out the results
        printf("total number of pattern matches found in chunk: %d\n", *host_count);
        printf("total characters processed: %d\n", total_chars_processed);
        printf("total patterns found: %d\n", total_count);
    }

    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("total execution time for 512MB file: %f", cpu_time_used);

    fclose(fp);

    // free up memory
    hipFree(text_dev);
    hipFree(pattern_dev);
    hipFree(summed_counts);
    hipFree(count);
    free(host_count);

    return 0;
}